#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define BLOCK_SIDE 16

typedef unsigned char ubyte;

__device__ ubyte getVal(ubyte* pboard, int rows, int columns, int x, int y)
{
    if (x >= 0 && x < columns && y >= 0 && y < rows)
        return pboard[x * columns + y];
    return 0x0;
}

__device__ int getNeighbours(int x, int y, ubyte* pboard, int rows, int columns)
{
    int num = 0;
    num += (getVal(pboard, rows, columns, x-1, y));
    num += (getVal(pboard, rows, columns, x+1, y));
    num += (getVal(pboard, rows, columns, x, y-1));
    num += (getVal(pboard, rows, columns, x, y+1));
    num += (getVal(pboard, rows, columns, x-1, y-1));
    num += (getVal(pboard, rows, columns, x-1, y+1));
    num += (getVal(pboard, rows, columns, x+1, y-1));
    num += (getVal(pboard, rows, columns, x+1, y+1));
    return num;
}

__global__ void step(int rows, int columns, ubyte* pCurrBoard, ubyte* pNewBoard)
{
    int x = blockIdx.x * BLOCK_SIDE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIDE + threadIdx.y;

    int index = x * columns + y;

    pNewBoard[index] = pCurrBoard[index];

    int neighbors = getNeighbours(x, y, pCurrBoard, rows, columns);

    // "Any live cell with fewer than two live neighbours dies as if caused by under-population."
    if (neighbors < 2)
        pNewBoard[index] = 0x0;

    // "Any live cell with more than three live neighbours dies, as if by overcrowding."
    if (neighbors > 3)
        pNewBoard[index] = 0x0;

    // "Any dead cell with exactly three live neighbours become a live cell, as if by reproduction."
    if (neighbors == 3 && !pCurrBoard[index])
        pNewBoard[index] = 0x1;
}

void generate(ubyte* pboard, int rows, int columns)
{
    for (int x = 0; x < columns; x++)
    {
        for (int y = 0; y < rows; y++)
        {
            float rnd = rand() / (float)RAND_MAX;
            pboard[x * columns + y] = (rnd >= 0.6f)? 0x1 : 0x0;
        }
    }
}

void printBoard(const char* msg, ubyte* pboard, int rows, int columns)
{
    printf("%s\n", msg);

    for (int x = 0; x < columns; x++)
    {
        for (int y = 0; y < rows; y++)
        {
            printf("%c ", pboard[x * columns + y]? 'o' : ' ');
        }
        printf("\n");
    }

}

int main(int argc, char* argv[])
{
    int generations = 1000000;
    int width = 100;
    int height = 100;

    if (argc > 1)
    {
        generations = atoi(argv[1]);
    }

    printf("Running %d generations\n", generations);

    srand(time(0));

    ubyte* pboard = (ubyte *)malloc(width * height * sizeof(ubyte));
    generate(pboard, height, width);
    printBoard("Initial Board:", pboard, height, width);

    ubyte* board;
    ubyte* newBoard;
    
    hipMalloc((void **)&board, width * height * sizeof(ubyte));
    hipMemcpy(board, pboard, height * width * sizeof(ubyte), hipMemcpyHostToDevice);
    hipMalloc((void **)&newBoard, width * height * sizeof(ubyte));
    hipMemset(newBoard, 0x0, height * width * sizeof(ubyte));

    dim3 blocksize(BLOCK_SIDE, BLOCK_SIDE);
    dim3 gridsize(width / BLOCK_SIDE, height / BLOCK_SIDE);

    struct timeval start;
    gettimeofday(&start, NULL);

    ubyte* pcurr;
    ubyte* pnext;
    for (int gen = 0; gen < generations; gen++){
        if ((gen % 2) == 0)
        {
            pcurr = board;
            pnext = newBoard;
        }
        else
        {
            pcurr = newBoard;
            pnext = board;
        }

        step<<<gridsize, blocksize>>>(height, width, pcurr, pnext);

#ifdef PRINT_BOARDS
        hipMemcpy(pboard, pnext, height * width * sizeof(ubyte), hipMemcpyDeviceToHost);
		for (int i = 0; i < 24; i++) printf("\n");
		printBoard(" ", pboard, height, width);
#endif
    }

    struct timeval end;
    gettimeofday(&end, NULL);
    double t =  (end.tv_usec - start.tv_usec) / 1000.0;

    hipMemcpy(pboard, pcurr, width * height * sizeof(ubyte), hipMemcpyDeviceToHost);

    printBoard("Board:", pboard, height, width);
    hipFree(board);
    hipFree(newBoard);
    free(pboard);

    printf("Computed %d generations in %f ms\n", generations, t);

    return 0;
}